#include "hip/hip_runtime.h"
#include "..\inc\knn_cuda.hpp"
#include <iostream>
#include <vector>
#include <cmath>
#include <map>
#include <fstream>
#include <sstream>

__global__ void KNN_CUDA(float *deviceData, float *testData, int rows, int cols, float *distances) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < rows) {
        float dist = 0.0;
        for (int j = 0; j < cols; ++j) {
            float diff = testData[j] - deviceData[tid * cols + j];
            dist += diff * diff;
        }
        distances[tid] = sqrt(dist);
    }
}

knn::KNN::KNN(const std::string pathToCSV, int k_NEIG, bool is_cudaTrue, int distanceType)
    : is_cudaTrue(is_cudaTrue), distanceType(distanceType), pathToCSV(pathToCSV), k_NEIG(k_NEIG) {
    hostData = csvTOvector(pathToCSV);
}

int knn::KNN::predict(const std::string testData) {
    std::vector<std::vector<float>> readyTestData = csvTOvector(testData);
    if (readyTestData.empty()) return -1;

    int row = readyTestData.size();
    int col = readyTestData[0].size();
    std::vector<float> h_testData(readyTestData[0].begin(), readyTestData[0].end());
    
    float *d_testData, *distances,*d_trainData;
    hipMalloc(&d_testData, row *col * sizeof(float));
    hipMalloc(&d_trainData,rows*cols* sizeof(float));
    hipMemcpy(d_testData, h_testData.data(), col * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_trainData, h_testData.data(), col * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&distances, rows * sizeof(float));
    int threads = 128;
    int blocks = (rows + threads - 1) / threads;

    KNN_CUDA<<<blocks, threads>>>(d_trainData, d_testData, rows, cols, distances);
    hipDeviceSynchronize();

    std::vector<float> h_distances(rows);
    hipMemcpy(h_distances.data(), distances, rows * sizeof(float), hipMemcpyDeviceToHost);

    int prediction = majorityCOUNT(h_distances.data());

    hipFree(d_testData);
    hipFree(distances);

    return prediction;
}

void knn::KNN::fit() {
    if (is_cudaTrue) {
        transferDataToDevice();
    } else {
        std::cout << "CUDA is disabled. Running on CPU." << std::endl;
    }
}

void knn::KNN::transferDataToDevice() {
    rows = hostData.size();
    cols = rows > 0 ? hostData[0].size() : 0;

    std::vector<float> h_data_temp(rows * cols);
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            h_data_temp[i * cols + j] = hostData[i][j];
        }
    }

    hipMalloc(&deviceData, rows * cols * sizeof(float));
    hipMemcpy(deviceData, h_data_temp.data(), rows * cols * sizeof(float), hipMemcpyHostToDevice);
}

int knn::KNN::majorityCOUNT(float* distances) {
    std::map<float, int> combinedDISLAB;
    for (int i = 0; i < labels.size(); ++i) {
        combinedDISLAB[distances[i]] = labels[i];
    }

    int ret = combinedDISLAB.begin()->second;
    int count = 1;
    std::map<int, int> occurCOUNT;
    occurCOUNT[ret] = 1;

    auto itr = combinedDISLAB.begin();
    for (int lab = 0; itr != combinedDISLAB.end() && lab < k_NEIG; ++itr, ++lab) {
        occurCOUNT[itr->second]++;
        if (occurCOUNT[itr->second] > count) {
            count = occurCOUNT[itr->second];
            ret = itr->second;
        }
    }
    return ret;
}

std::vector<std::vector<float>> knn::KNN::csvTOvector(const std::string path) {
    std::ifstream file(path);
    std::string line;
    std::vector<std::vector<float>> ret;

    if (file.is_open()) {
        while (getline(file, line)) {
            std::stringstream ss(line);
            std::string value;
            std::vector<float> row;
            while (getline(ss, value, ',')) {
                row.push_back(std::stof(value));
            }
            ret.push_back(std::vector<float>(row.begin(), row.end() - 1));
            labels.push_back(row.back());
        }
        file.close();
    } else {
        std::cerr << "Unable to open file" << std::endl;
    }

    return ret;
}
