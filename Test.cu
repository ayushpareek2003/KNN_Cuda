#include <iostream>
#include <vector>
#include <fstream>
#include <sstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
__global__ void KNN_CUDA(float *deviceData, float *testData, int rows, int cols, float *distances) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < rows*cols) {
        int row = tid / cols; 
        int col = tid % cols; 
        
        float diff = deviceData[tid] - testData[col]; 
        
        atomicAdd(&distances[row], diff * diff); 

    }
}




int main(){
    float arr[] = {5.0, 5.8, 4.6, 1.2, 1.5,7.8};

    float *d;
    hipMalloc(&d,6*sizeof(float));
    hipMemcpy(d,arr,6*sizeof(float),hipMemcpyHostToDevice);

    float tes[]={5.0,6.7};

    float *de;
    hipMalloc(&de,2*sizeof(float));
    hipMemcpy(de,tes,2*sizeof(float),hipMemcpyHostToDevice);

    float *dis;
    hipMalloc(&dis,3*sizeof(float));

    KNN_CUDA<<<1,128>>>(d,de,3,2,dis);

    float *arr2;
    arr2=(float*)malloc(3*sizeof(float));
    hipMemcpy(arr2,dis,3*sizeof(float),hipMemcpyDeviceToHost);

    for(int l=0;l<3;l++){
        std::cout<<arr2[l]<<" ";
    }

    return 0;







}