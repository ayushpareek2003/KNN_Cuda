#include "hip/hip_runtime.h"
#include "..\inc\knn_cuda.hpp"
#include <hip/hip_runtime.h>




__global__ int KNN_CUDA(float **deviceData,int rows,int col,float **deviceDataNew){

    int x=threadIdx.x+blockDim.x*blockIdx.x;

    
    





}

knn::KNN::KNN(bool is_cudaTrue,int distanceType,const std::string pathToCSV):is_cudaTrue(is_cudaTrue),
                                                            distanceType(distanceType),
                                                            pathToCSV(pathToCSV){

    hostData=csvTOvector(pathToCSV);

}



int knn::KNN::predict(const std::vector<std::vector<float>> newData){

    



}

void knn::KNN::fit(){
    float* deviceData;
    if(is_cudaTrue){
        transferDataToDevice();

    }
    else{
        std::cout<<"USE CUDA BHAI"<<std::endl;

        
    }


}

void knn::KNN::transferDataToDevice(){

        int rows=hostData.size();
        int col=0;
        if(rows>0){
            col=hostData[0].size();
        }
        else{
            col=0;
        }
        float* h_data_temp=new float[rows*col];

        for(int i=0;i<rows;i++){
            for(int j=0;j<col;j++){
                h_data_temp[i*(col)+j]=hostData[i][j];

            }
        }

        //cuda time 
        hipMalloc(deviceData,sizeof(float)*rows*col);
        hipMemcpy(deviceData,h_data_temp,sizeof(float)*rows*col,hipMemcpyHostToDevice);

        delete[] h_data_temp;

}

std::vector<std::vector<float>> knn::KNN::csvTOvector(const std::string path){
            std::ifstream file(pathToCSV);
            std::string line;

            std::vector<std::vector<float>> ret;
            if(file.is_open()){
                while(getline(file,line)){
                    std::stringstream ss(line);
                    std::string value;
                    std::vector<float> row;
                    while(getline(ss,value,',')){
                        row.push_back(std::stof(value));
                    }
                    ret.push_back(row);
                }
                file.close();
            }  
            else{
                std::cerr<<"Unable to open file"<<std::endl;
                
            } 

            return ret;
}


