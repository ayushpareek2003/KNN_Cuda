#include "hip/hip_runtime.h"
#include "..\inc\knn_cuda.hpp"



__constant__ float* newData[128];

__global__ void knn::KNN::KNN_CUDA(float **deviceData,int rows,int cols,float *distances){


    int r=threadIdx.x;
    int c=blockDim.x*blockIdx.x;

    __shared__ float distancesTEMPblock[128];

    if(r+c<rows*cols){
        float* a_TEMP=newData[r];
        float* b_TEMP=deviceData[r+c];
        distancesTEMPblock[r/cols]+=(a_TEMP-b_TEMP)*(a_TEMP-b_TEMP);
        __syncthreads();
    }

    distances[r/cols]=distancesTEMPblock[r/cols];
    

}

knn::KNN::KNN(bool is_cudaTrue,int distanceType,const std::string pathToCSV):is_cudaTrue(is_cudaTrue),
                                                            distanceType(distanceType),
                                                            pathToCSV(pathToCSV){

    hostData=csvTOvector(pathToCSV);

}



int knn::KNN::predict(const std::string testData){

    std::vector<std::vector<float>> readyTestData=csvTOvector(testData);

    int row=readyTestData.size();
    int col=0;
    if(row>0){
            col=readyTestData[0].size();
    }
    else if(row>1){
        return 0;
            col=0;
    }
    float* h_data_temp=new float[row*col];

    for(int i=0;i<row;i++){
        for(int j=0;j<col;j++){
            h_data_temp[i*(col)+j]=hostData[i][j];

        }
    }

    hipMemcpyToSymbol(HIP_SYMBOL(newData),h_data_temp,sizeof(h_data_temp));

    float *distances;
    distances=(float*)malloc(sizeof(float)*rows);

    dim3 blocks=(ceil((row*col)/128));
    dim3 threads=(128);

    knn::KNN::KNN_CUDA<<<blocks,threads>>>(deviceData,rows,cols,distances);

    //code to answer that soon i will write that// 











}

void knn::KNN::fit(){
    float* deviceData;
    if(is_cudaTrue){
        transferDataToDevice();

    }
    else{
        std::cout<<"USE CUDA BHAI"<<std::endl;

        
    }


}

void knn::KNN::transferDataToDevice(){

        rows=hostData.size();
        cols=0;
        if(rows>0){
            cols=hostData[0].size();
        }
        else{
            cols=0;
        }
        float* h_data_temp=new float[rows*cols];

        for(int i=0;i<rows;i++){
            for(int j=0;j<cols;j++){
                h_data_temp[i*(cols)+j]=hostData[i][j];

            }
        }

        //cuda time 
        
        hipMalloc(deviceData,sizeof(float)*rows*cols);
        hipMemcpy(deviceData,h_data_temp,sizeof(float)*rows*cols,hipMemcpyHostToDevice);

        delete[] h_data_temp;

}

std::vector<std::vector<float>> knn::KNN::csvTOvector(const std::string path){
            std::ifstream file(pathToCSV);
            std::string line;

            std::vector<std::vector<float>> ret;
            if(file.is_open()){
                while(getline(file,line)){
                    std::stringstream ss(line);
                    std::string value;
                    std::vector<float> row;
                    while(getline(ss,value,',')){
                        row.push_back(std::stof(value));
                    }
                    ret.push_back(row);
                }
                file.close();
            }  
            else{
                std::cerr<<"Unable to open file"<<std::endl;
                
            } 

            return ret;
}


