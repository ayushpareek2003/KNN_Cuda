#include "hip/hip_runtime.h"
#include "knn_cuda.cuh"

//######################## Ayush Pareek ##################################### (07-03-2025)


//////////////////////////////////////////////////
////////// This is more optimised code ///////////
//////////////////////////////////////////////////


// __constant__ float* newData[128];

// __global__ void KNN_CUDA(float **deviceData,int rows,int cols,float *distances){


//     int r=threadIdx.x;
//     int c=blockDim.x*blockIdx.x;

//     __shared__ float distancesTEMPblock[128];

//     if(r+c<rows*cols){
//         float* a_TEMP=newData[r];
//         float* b_TEMP=deviceData[r+c];
//         distancesTEMPblock[r/cols]+=(a_TEMP-b_TEMP)*(a_TEMP-b_TEMP);
//         __syncthreads();
//     }

//        if (r < 128 && c < rows) {
//         distances[r/cols] = sqrtf(distancesTEMPblock[r/cols]);  // Store the result in distances
//     }

// }



//////////////////////////////////////////////////
///////////i will make it work one day ///////////
//////////////////////////////////////////////////



/// temporary function 
__global__ void KNN_CUDA(float *deviceData, float *testData, int rows, int cols, float *distances) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < rows*cols) {
       
        int row = tid / cols; 
        int col = tid % cols; 
        
        float diff = deviceData[tid] - testData[col]; 
     
        atomicAdd(&distances[row], diff * diff); 

    }
}

knn::KNN::KNN(const std::string pathToCSV,int k_NEIG,bool is_cudaTrue,int distanceType):is_cudaTrue(is_cudaTrue),
                                                            distanceType(distanceType),
                                                            pathToCSV(pathToCSV),k_NEIG(k_NEIG){

    hostData=csvTOvector(pathToCSV);

}



int knn::KNN::predict(const std::string testData){

    std::vector<std::vector<float>> readyTestData=csvTOvector(testData);

    int row=readyTestData.size();
    int col=readyTestData[0].size();

    std::cout<<row<<" "<<col;

    for(int l=0;l<row;l++){
        if(readyTestData[l].size()!=cols){
            std::cout<<cols<<readyTestData[0].size();
            std::cerr<<"No of columns dont match"<<std::endl;
            return -1;
        }
    }

    float* h_data_temp;
    h_data_temp=(float*)malloc(sizeof(float)*row*col);

    for(int i=0;i<row;i++){
        for(int j=0;j<col;j++){
            h_data_temp[i*(col)+j]=readyTestData[i][j];

        }
    }

    float *newData_GPU;

    // hipMemcpyToSymbol(HIP_SYMBOL(newData),h_data_temp,sizeof(h_data_temp));
    hipMalloc(&newData_GPU,row*col*sizeof(float));
    hipMemcpy(newData_GPU,h_data_temp,row*col*sizeof(float),hipMemcpyHostToDevice);


    float *distances;
    hipMalloc(&distances,rows*sizeof(float));

    dim3 blocks=(ceil((row*col)/128));
    dim3 threads=(128);
 
    KNN_CUDA<<<1,128>>>(deviceData,newData_GPU,rows,cols,distances);
    hipDeviceSynchronize();

    //code to answer that soon i will write that// 
    float *distances_host;
    distances_host=(float*)malloc(sizeof(float)*rows);
    
    hipMemcpy(distances_host,distances,sizeof(float)*rows,hipMemcpyDeviceToHost);
    
    hipFree(distances);
    
    int prediction=majorityCOUNT(distances_host);

    return prediction;
    
    
}

void knn::KNN::fit(){
    float* deviceData;
    if(is_cudaTrue){
        transferDataToDevice();

    }
    else{
        std::cout<<"USE CUDA BHAI"<<std::endl;
    }


}

void knn::KNN::transferDataToDevice(){

        rows=hostData.size();
        cols=0;
        if(rows>0){
            cols=hostData[0].size();
        }
        else{
            cols=0;
        }
        float* h_data_temp=new float[rows*cols];

        for(int i=0;i<rows;i++){
            for(int j=0;j<cols;j++){
                h_data_temp[i*(cols)+j]=hostData[i][j];

            }
        }

        //cuda time 
        
        hipMalloc(&deviceData,sizeof(float)*rows*cols);
        hipMemcpy(deviceData,h_data_temp,sizeof(float)*rows*cols,hipMemcpyHostToDevice);

        delete[] h_data_temp;

}


int knn::KNN::majorityCOUNT(float* distances){
    std::map<float,int> combinedDISLAB;

    // std::cout<<labels.size()<<std::endl;
    for(int i=0;i<labels.size();i++){
        combinedDISLAB[distances[i]]=labels[i];

        // std::cout<<distances[i]<<" "<<labels[i]<<std::endl;
    }

    auto itr=combinedDISLAB.begin();

    int ret=itr->second; //default case when no ones get majority , too sbsey kum distance walla return krunga
    float dist=itr->first;
    int count=1;
    std::map<int,int> occurCOUNT;
    occurCOUNT[ret]=1;
    int lab=1;
    itr++;

    while(itr!=combinedDISLAB.end() && lab<k_NEIG){
        if(occurCOUNT.find(itr->second)==occurCOUNT.end()){
            occurCOUNT[itr->second]=1;
        }
        else{
            occurCOUNT[itr->second]+=1;
        }

        if(count<occurCOUNT[itr->second]){
            count=occurCOUNT[itr->second];
            ret=itr->second;
            
        }
        itr++;
        lab++;

    }

    return ret;

}

std::vector<std::vector<float>> knn::KNN::csvTOvector(const std::string path){
            std::ifstream file(path);
            std::string line;

            std::vector<std::vector<float>> ret; //keeping my legacy of naming returning variable as ret (LEETCODE se sikhey h 22)
            if(file.is_open()){
                while(getline(file,line)){
                    std::stringstream ss(line);
                    std::string value;
                    std::vector<float> row;
                    while(getline(ss,value,',')){
                        row.push_back(std::stof(value));

                    }

                    
                    ret.push_back(std::vector<float>(row.begin(),row.end()-1));
                   
                    labels.push_back(row[row.size()-1]);
                }
                file.close();
            }  
            else{
                std::cerr<<"Unable to open file"<<std::endl;
                
            }
            // ret.pop_back();
            // labels.pop_back();
            // for(auto k:labels){
            //     std::cout<<k<<" ";
            // }

            return ret;
}


